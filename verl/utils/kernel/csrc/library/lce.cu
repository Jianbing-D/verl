#include "hip/hip_runtime.h"
#include "interfaces.h"
#include "sm80.h"

namespace lce {

template <>
void forward_mainloop<float, float>(int32_t rank,
                                    void *hidden_ptr,
                                    void *weight_ptr,
                                    int64_t *labels_ptr,
                                    int32_t num_tokens,
                                    int32_t vocab_size,
                                    int32_t vocab_per_split,
                                    void *max_ptr,
                                    void *acc_ptr,
                                    void *entropy_b_ptr,
                                    void *logprobs_ptr,
                                    float *gmem_output_ptr,
                                    hipStream_t stream) {}

template <>
void forward_mainloop<__hip_bfloat16, __hip_bfloat16>(int32_t rank,
                                                    void *hidden_ptr,
                                                    void *weight_ptr,
                                                    int64_t *labels_ptr,
                                                    int32_t num_tokens,
                                                    int32_t vocab_size,
                                                    int32_t vocab_per_split,
                                                    void *max_ptr,
                                                    void *acc_ptr,
                                                    void *entropy_b_ptr,
                                                    void *logprobs_ptr,
                                                    float *gmem_output_ptr,
                                                    hipStream_t stream) {
    // first, lets check whether the GEMM is correct
    using Traits = lce::Traits<__hip_bfloat16, __hip_bfloat16, 4096>;

    int32_t num_splits = (vocab_size + vocab_per_split - 1) / vocab_per_split;
    int32_t last_split_size = vocab_size - (num_splits - 1) * vocab_per_split;
    if (last_split_size % 4 != 0) {
        // NOTE: such requirement is due to the GMEM vectorized store
        throw std::invalid_argument("last split size must be divisible by 4 for address alignment");
    }

    // thread-block swizzle
    int32_t num_blocks = (num_tokens + Traits::tileM - 1) / Traits::tileM;
    num_blocks *= Traits::threadBlockSwizzleSize;

    num_blocks *= ((num_splits + Traits::threadBlockSwizzleSize - 1) / Traits::threadBlockSwizzleSize);

    dim3 block(Traits::threads, 1, 1);
    dim3 grid(num_blocks, 1, 1);

    auto kernel = lce::forward_mainloop_kernel<Traits>;
    if (Traits::smem_bytes >= 48 * 1024ul) {
        CUDA_THROW(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             Traits::smem_bytes));
    }

#if 0
    // print_latex(Traits::SmemLayoutAtom{});
    print_latex(Traits::SmemLayoutHidden{});
    printf("block: (%d, %d, %d), grid: (%d, %d, %d), smem_bytes: %ld\n",
        block.x, block.y, block.z, grid.x, grid.y, grid.z, Traits::smem_bytes);
#endif

    kernel<<<grid, block, Traits::smem_bytes, stream>>>(
        rank,
        reinterpret_cast<typename Traits::IN_DTYPE*>(hidden_ptr),
        reinterpret_cast<typename Traits::IN_DTYPE*>(weight_ptr),
        labels_ptr,
        num_tokens,
        vocab_size,
        vocab_per_split,
        num_splits,
        reinterpret_cast<float*>(max_ptr),
        reinterpret_cast<float*>(acc_ptr),
        reinterpret_cast<float*>(entropy_b_ptr),
        reinterpret_cast<float*>(logprobs_ptr),
        reinterpret_cast<float*>(gmem_output_ptr)
    );
    CUDA_THROW(hipGetLastError());
}

} // namespace lce