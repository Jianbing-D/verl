#include "hip/hip_runtime.h"
#include "interfaces.h"
#include "sm80.h"

namespace lce {

template <>
void forward_mainloop<float, float>(
                    int32_t rank,
                    void *hidden_ptr,
                    int32_t stride_hidden_m, int32_t stride_hidden_k,
                    void *weight_ptr,
                    int32_t stride_weight_n, int32_t stride_weight_k,
                    uint64_t *labels_ptr,
                    int32_t num_tokens,
                    int32_t vocab_size,
                    int32_t vocab_per_split,
                    float *gmem_output_ptr,
                    hipStream_t stream) {
}

template <>
void forward_mainloop<__hip_bfloat16, __hip_bfloat16>(
                    int32_t rank,
                    void *hidden_ptr,
                    int32_t stride_hidden_m, int32_t stride_hidden_k,
                    void *weight_ptr,
                    int32_t stride_weight_n, int32_t stride_weight_k,
                    uint64_t *labels_ptr,
                    int32_t num_tokens,
                    int32_t vocab_size,
                    int32_t vocab_per_split,
                    float *gmem_output_ptr,
                    hipStream_t stream) {
    // first, lets check whether the GEMM is correct
    using Traits = lce::Traits<__hip_bfloat16, __hip_bfloat16, 4096>;

    int32_t num_blocks = (num_tokens + Traits::tileM - 1) / Traits::tileM;
    dim3 block(Traits::threads, 1, 1);
    dim3 grid(num_blocks, 1, 1);

    // printf("block: (%d, %d, %d), grid: (%d, %d, %d), smem_bytes: %ld\n",
    //        block.x, block.y, block.z, grid.x, grid.y, grid.z, Traits::smem_bytes);

    auto kernel = lce::forward_mainloop_kernel<Traits>;
    if (Traits::smem_bytes >= 48 * 1024ul) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(kernel),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             Traits::smem_bytes);
    }

#if 0
    // print_latex(Traits::SmemLayoutAtom{});
    print_latex(Traits::SmemLayoutHidden{});
#endif

    kernel<<<grid, block, Traits::smem_bytes, stream>>>(
        rank,
        reinterpret_cast<typename Traits::IN_DTYPE*>(hidden_ptr),
        stride_hidden_m, stride_hidden_k,
        reinterpret_cast<typename Traits::IN_DTYPE*>(weight_ptr),
        stride_weight_n, stride_weight_k,
        labels_ptr,
        num_tokens,
        vocab_size,
        vocab_per_split,
        reinterpret_cast<float*>(gmem_output_ptr)
    );
}

} // namespace lce